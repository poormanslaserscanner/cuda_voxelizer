#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <iostream>
#include "util_cuda.h"
#include "util_common.h"

// CUDA Global Memory variables
//__device__ size_t voxel_count = 0; // How many voxels did we count
//__device__ size_t triangles_seen_count = 0; // Sanity check

__constant__ uint32_t morton256_x[256];
__constant__ uint32_t morton256_y[256];
__constant__ uint32_t morton256_z[256];

// Encode morton code using LUT table
__device__ inline uint64_t mortonEncode_LUT(unsigned int x, unsigned int y, unsigned int z){
	uint64_t answer = 0;
	answer = morton256_z[(z >> 16) & 0xFF] |
		morton256_y[(y >> 16) & 0xFF] |
		morton256_x[(x >> 16) & 0xFF];
	answer = answer << 48 |
		morton256_z[(z >> 8) & 0xFF] |
		morton256_y[(y >> 8) & 0xFF] |
		morton256_x[(x >> 8) & 0xFF];
	answer = answer << 24 |
		morton256_z[(z)& 0xFF] |
		morton256_y[(y)& 0xFF] |
		morton256_x[(x)& 0xFF];
	return answer;
}

// Possible optimization: buffer bitsets (for now: too much overhead)
struct bufferedBitSetter{
	unsigned int* voxel_table;
	size_t current_int_location;
	unsigned int current_mask;

	__device__ __inline__ bufferedBitSetter(unsigned int* voxel_table, size_t index) :
		voxel_table(voxel_table), current_mask(0) {
		current_int_location = index / size_t(32);
	}

	__device__ __inline__ void setBit(size_t index){
		size_t new_int_location = index / size_t(32);
		if (current_int_location != new_int_location){
			flush();
			current_int_location = new_int_location;
		}
		unsigned int bit_pos = 31 - (unsigned int)(int(index) % 32);
		current_mask = current_mask | (1 << bit_pos);
	}

	__device__ __inline__ void flush(){
		if (current_mask != 0){
			atomicOr(&(voxel_table[current_int_location]), current_mask);
		}
	}
};

__device__ __inline__ bool checkBit(unsigned int* voxel_table, size_t index){
	size_t int_location = index / size_t(32);
	unsigned int bit_pos = size_t(31) - (index % size_t(32)); // we count bit positions RtL, but array indices LtR
	return ((voxel_table[int_location]) & (1 << bit_pos));
}

__device__ __inline__ void setBit(unsigned int* voxel_table, size_t index){
	size_t int_location = index / size_t(32);
	unsigned int bit_pos = size_t(31) - (index % size_t(32)); // we count bit positions RtL, but array indices LtR
	unsigned int mask = 1 << bit_pos;
	atomicOr(&(voxel_table[int_location]), mask);
}

// Main triangle voxelization method
__global__ void voxelize_triangle(voxinfo info, float* triangle_data, unsigned int* voxel_table, bool morton_order){
	size_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;

	// Common variables
	glm::vec3 delta_p = glm::vec3(1.0, 1.0, 1.0);
	glm::vec3 c(0.0f, 0.0f, 0.0f); // critical point
	glm::vec3 world_base(glm::vec3(info.bbox.min));
	size_t gridsiz2 = info.gridsize*info.gridsize;
	while (thread_id < info.n_triangles){ // every thread works on specific triangles in its stride
		size_t t = thread_id * 9; // triangle contains 9 vertices

		// COMPUTE COMMON TRIANGLE PROPERTIES
		glm::vec3 v0 = glm::vec3(triangle_data[t + 0], triangle_data[t + 1], triangle_data[t + 2]) - world_base; // get v0 and move to origin
		glm::vec3 v1 = glm::vec3(triangle_data[t + 3], triangle_data[t + 4], triangle_data[t + 5]) - world_base; // get v1 and move to origin
		glm::vec3 v2 = glm::vec3(triangle_data[t + 6], triangle_data[t + 7], triangle_data[t + 8]) - world_base; // get v2 and move to origin
		glm::vec3 e0 = v1 - v0;
		glm::vec3 e1 = v2 - v1;
		glm::vec3 e2 = v0 - v2;
		glm::vec3 n = glm::normalize(glm::cross(e0, e1));

		//COMPUTE TRIANGLE BBOX IN GRID
		AABox<glm::vec3> t_bbox_world(glm::min(v0, glm::min(v1, v2)), glm::max(v0, glm::max(v1, v2)));
		AABox<glm::ivec3> t_bbox_grid;
		t_bbox_grid.min = glm::clamp(glm::ivec3(floor(t_bbox_world.min)),
			glm::ivec3(1, 1, 1), 
			glm::ivec3(int(info.gridsize-1), int(info.gridsize-1), int(info.gridsize-1)));
		t_bbox_grid.max = glm::clamp(glm::ivec3(ceil(t_bbox_world.max)),
			glm::ivec3(1, 1, 1), 
			glm::ivec3(int(info.gridsize-1), int(info.gridsize-1), int(info.gridsize-1)));
//		t_bbox_grid.min = floor(t_bbox_world.min / info.unit);
//		t_bbox_grid.max = ceil(t_bbox_world.max / info.unit);

		// PREPARE PLANE TEST PROPERTIES
		if (n.x > 0.0f) { c.x = 1.0; }
		if (n.y > 0.0f) { c.y = 1.0; }
		if (n.z > 0.0f) { c.z = 1.0; }
		float d1 = glm::dot(n, (c - v0));
		float d2 = glm::dot(n, ((delta_p - c) - v0));

		// PREPARE PROJECTION TEST PROPERTIES
		// XY plane
		glm::vec2 n_xy_e0(-1.0f*e0.y, e0.x);
		glm::vec2 n_xy_e1(-1.0f*e1.y, e1.x);
		glm::vec2 n_xy_e2(-1.0f*e2.y, e2.x);
		if (n.z < 0.0f) {
			n_xy_e0 = -n_xy_e0;
			n_xy_e1 = -n_xy_e1;
			n_xy_e2 = -n_xy_e2;
		}
		float d_xy_e0 = (-1.0f * glm::dot(n_xy_e0, glm::vec2(v0.x, v0.y))) + glm::max(0.0f, n_xy_e0[0]) + glm::max(0.0f, n_xy_e0[1]);
		float d_xy_e1 = (-1.0f * glm::dot(n_xy_e1, glm::vec2(v1.x, v1.y))) + glm::max(0.0f, n_xy_e1[0]) + glm::max(0.0f, n_xy_e1[1]);
		float d_xy_e2 = (-1.0f * glm::dot(n_xy_e2, glm::vec2(v2.x, v2.y))) + glm::max(0.0f, n_xy_e2[0]) + glm::max(0.0f, n_xy_e2[1]);
		// YZ plane
		glm::vec2 n_yz_e0(-1.0f*e0.z, e0.y);
		glm::vec2 n_yz_e1(-1.0f*e1.z, e1.y);
		glm::vec2 n_yz_e2(-1.0f*e2.z, e2.y);
		if (n.x < 0.0f) {
			n_yz_e0 = -n_yz_e0;
			n_yz_e1 = -n_yz_e1;
			n_yz_e2 = -n_yz_e2;
		}
		float d_yz_e0 = (-1.0f * glm::dot(n_yz_e0, glm::vec2(v0.y, v0.z))) + glm::max(0.0f, n_yz_e0[0]) + glm::max(0.0f, n_yz_e0[1]);
		float d_yz_e1 = (-1.0f * glm::dot(n_yz_e1, glm::vec2(v1.y, v1.z))) + glm::max(0.0f, n_yz_e1[0]) + glm::max(0.0f, n_yz_e1[1]);
		float d_yz_e2 = (-1.0f * glm::dot(n_yz_e2, glm::vec2(v2.y, v2.z))) + glm::max(0.0f, n_yz_e2[0]) + glm::max(0.0f, n_yz_e2[1]);
		// ZX plane
		glm::vec2 n_zx_e0(-1.0f*e0.x, e0.z);
		glm::vec2 n_zx_e1(-1.0f*e1.x, e1.z);
		glm::vec2 n_zx_e2(-1.0f*e2.x, e2.z);
		if (n.y < 0.0f) {
			n_zx_e0 = -n_zx_e0;
			n_zx_e1 = -n_zx_e1;
			n_zx_e2 = -n_zx_e2;
		}
		float d_xz_e0 = (-1.0f * glm::dot(n_zx_e0, glm::vec2(v0.z, v0.x))) + glm::max(0.0f, n_zx_e0[0]) + glm::max(0.0f, n_zx_e0[1]);
		float d_xz_e1 = (-1.0f * glm::dot(n_zx_e1, glm::vec2(v1.z, v1.x))) + glm::max(0.0f, n_zx_e1[0]) + glm::max(0.0f, n_zx_e1[1]);
		float d_xz_e2 = (-1.0f * glm::dot(n_zx_e2, glm::vec2(v2.z, v2.x))) + glm::max(0.0f, n_zx_e2[0]) + glm::max(0.0f, n_zx_e2[1]);

		// test possible grid boxes for overlap
		for (int z = t_bbox_grid.min.z - 1; z <= t_bbox_grid.max.z; z++){
			for (int y = t_bbox_grid.min.y - 1; y <= t_bbox_grid.max.y; y++){
				for (int x = t_bbox_grid.min.x - 1; x <= t_bbox_grid.max.x; x++){
					// size_t location = x + (y*info.gridsize) + (z*info.gridsize*info.gridsize);
					// if (checkBit(voxel_table, location)){ continue; }
					// TRIANGLE PLANE THROUGH BOX TEST
					glm::vec3 p((x + 0.5f), (y + 0.5f), (z + 0.5f));
					float nDOTp = glm::dot(n, p);
					if ((nDOTp + d1) * (nDOTp + d2) > 0.0f){ continue; }

					// PROJECTION TESTS
					// XY
					glm::vec2 p_xy(p.x, p.y);
					if ((glm::dot(n_xy_e0, p_xy) + d_xy_e0) < 0.0f){ continue; }
					if ((glm::dot(n_xy_e1, p_xy) + d_xy_e1) < 0.0f){ continue; }
					if ((glm::dot(n_xy_e2, p_xy) + d_xy_e2) < 0.0f){ continue; }

					// YZ
					glm::vec2 p_yz(p.y, p.z);
					if ((glm::dot(n_yz_e0, p_yz) + d_yz_e0) < 0.0f){ continue; }
					if ((glm::dot(n_yz_e1, p_yz) + d_yz_e1) < 0.0f){ continue; }
					if ((glm::dot(n_yz_e2, p_yz) + d_yz_e2) < 0.0f){ continue; }

					// XZ	
					glm::vec2 p_zx(p.z, p.x);
					if ((glm::dot(n_zx_e0, p_zx) + d_xz_e0) < 0.0f){ continue; }
					if ((glm::dot(n_zx_e1, p_zx) + d_xz_e1) < 0.0f){ continue; }
					if ((glm::dot(n_zx_e2, p_zx) + d_xz_e2) < 0.0f){ continue; }

					//atomicAdd(&voxel_count, 1);
					size_t location = size_t(x + (y*info.gridsize)) + (size_t(z)*gridsiz2);
					setBit(voxel_table, location);
					continue;
				}
			}
		}
		// sanity check: atomically count triangles
		//atomicAdd(&triangles_seen_count, 1);
		thread_id += stride;
	}
}

void voxelize(const voxinfo& v, float* triangle_data, unsigned int* vtable, bool morton_code){
	float   elapsedTime;

	// Create timers, set start time
	hipEvent_t start_total, stop_total, start_vox, stop_vox;
	HANDLE_CUDA_ERROR(hipEventCreate(&start_total));
	HANDLE_CUDA_ERROR(hipEventCreate(&stop_total));
	HANDLE_CUDA_ERROR(hipEventCreate(&start_vox));
	HANDLE_CUDA_ERROR(hipEventCreate(&stop_vox));
	HANDLE_CUDA_ERROR(hipEventRecord(start_total, 0));

	// Copy morton LUT if we're encoding to morton
	if (morton_code){
		HANDLE_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(morton256_x), host_morton256_x, 256 * sizeof(uint32_t)));
		HANDLE_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(morton256_y), host_morton256_y, 256 * sizeof(uint32_t)));
		HANDLE_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(morton256_z), host_morton256_z, 256 * sizeof(uint32_t)));
	}

	// Estimate best block and grid size using CUDA Occupancy Calculator
	int blockSize;   // The launch configurator returned block size 
	int minGridSize; // The minimum grid size needed to achieve the  maximum occupancy for a full device launch 
	int gridSize;    // The actual grid size needed, based on input size 
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, voxelize_triangle);
	// Round up according to array size 
	gridSize = int( (v.n_triangles + blockSize - 1) / blockSize );

	HANDLE_CUDA_ERROR(hipEventRecord(start_vox, 0));
	voxelize_triangle << <gridSize, blockSize >> >(v, triangle_data, vtable, morton_code);
	CHECK_CUDA_ERROR();

	hipDeviceSynchronize();
	HANDLE_CUDA_ERROR(hipEventRecord(stop_vox, 0));
	HANDLE_CUDA_ERROR(hipEventSynchronize(stop_vox));
	HANDLE_CUDA_ERROR(hipEventElapsedTime(&elapsedTime, start_vox, stop_vox));
	printf("Voxelisation GPU time:  %3.1f ms\n", elapsedTime);

	// SANITY CHECKS
	//size_t t_seen, v_count;
	//HANDLE_CUDA_ERROR(hipMemcpyFromSymbol((void*)&(t_seen),triangles_seen_count, sizeof(t_seen), 0, hipMemcpyDeviceToHost));
	//HANDLE_CUDA_ERROR(hipMemcpyFromSymbol((void*)&(v_count), voxel_count, sizeof(v_count), 0, hipMemcpyDeviceToHost));
	//printf("We've seen %llu triangles on the GPU \n", t_seen);
	//printf("We've found %llu voxels on the GPU \n", v_count);

	// get stop time, and display the timing results
	HANDLE_CUDA_ERROR(hipEventRecord(stop_total, 0));
	HANDLE_CUDA_ERROR(hipEventSynchronize(stop_total));
	HANDLE_CUDA_ERROR(hipEventElapsedTime(&elapsedTime, start_total, stop_total));
	printf("Total GPU time (including memory transfers):  %3.1f ms\n", elapsedTime);

	// Destroy timers
	HANDLE_CUDA_ERROR(hipEventDestroy(start_total));
	HANDLE_CUDA_ERROR(hipEventDestroy(stop_total));
	HANDLE_CUDA_ERROR(hipEventDestroy(start_vox));
	HANDLE_CUDA_ERROR(hipEventDestroy(stop_vox));
}
